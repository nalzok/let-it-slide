#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <vector>
#include <utility>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include <cuda_pipeline.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace torch::indexing;
using namespace nvcuda;

#define MAX_THREADS_PER_BLOCK 256

#define FULL_MASK 0xFFFFFFFFU
#define HALF_MASK 0x0000FFFFU

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}

typedef union __align__(4) alias_half_uint16 {
    half2 foo;
    uint16_t bar[2];
} alias_half_uint16;


__global__ static void
__launch_bounds__(MAX_THREADS_PER_BLOCK)
decompress_kernel(
    half *__restrict__ out,
    const uint4 *__restrict__ compressed,
    const half *__restrict__ codebook,
    size_t compressed_m,
    size_t compressed_n
) {
    size_t laneId = threadIdx.x % warpSize;
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    for (int elem_idx = threadId; elem_idx < compressed_m * compressed_n; elem_idx += gridDim.x * blockDim.x) {
        uint4 inputs = compressed[elem_idx];
        uint16_t state = __shfl_up_sync(FULL_MASK, inputs.w, 1);
        if (laneId == 0) {
            if (elem_idx % compressed_n == 0) {
                // first element in a row
                state = 0;
            } else {
                const uint16_t *ptr = reinterpret_cast<const uint16_t *>(compressed + elem_idx);
                state = ptr[-1];
            }
        }

        half *__restrict__ output = out + elem_idx * 64;

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.x, state, 2);
            inputs.x <<= 2;
            output[i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.y, state, 2);
            inputs.y <<= 2;
            output[16+i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.z, state, 2);
            inputs.z <<= 2;
            output[32+i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.w, state, 2);
            inputs.w <<= 2;
            output[48+i] = codebook[state];
        }
    }
}


__host__ extern float decompress(
    torch::Tensor &compressed,
    torch::Tensor &codebook,
    torch::Tensor &out
) {
    CHECK_INPUT(compressed);
    TORCH_CHECK(compressed.dim() == 2);
    TORCH_CHECK(compressed.size(1) % (4 * 32) == 0);    // each warp has 32 threads, each handling an uint4
    TORCH_CHECK(compressed.scalar_type() == torch::kInt32);

    size_t compressed_m = compressed.size(0);
    size_t compressed_n = compressed.size(1) / 4;
    size_t m = compressed_m;
    size_t n = compressed_n * 64;   // at 2 bit, each uint4 has 4x32 bits = 4x16 weights

    CHECK_INPUT(codebook);
    TORCH_CHECK(codebook.dim() == 1);
    TORCH_CHECK(codebook.size(0) == 1<<16);
    TORCH_CHECK(codebook.scalar_type() == torch::kFloat16);

    CHECK_INPUT(out);
    TORCH_CHECK(out.dim() == 2);
    TORCH_CHECK(out.size(0) == m);
    TORCH_CHECK(out.size(1) == n);
    TORCH_CHECK(out.scalar_type() == torch::kFloat16);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, compressed.get_device());
    size_t grid_size = 2 * static_cast<size_t>(deviceProp.multiProcessorCount);
    size_t block_size = MAX_THREADS_PER_BLOCK;

    hipStream_t stream;
    gpuErrchk(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipEventRecord(start, stream));

    decompress_kernel<<<grid_size, block_size, 0, stream>>>(
        (half *)out.data_ptr<c10::Half>(),
        (const uint4 *)compressed.data_ptr<int32_t>(),
        (const half *)codebook.data_ptr<c10::Half>(),
        compressed_m,
        compressed_n);
    
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return msecTotal;
}


template <size_t L>
__global__ static void
__launch_bounds__(MAX_THREADS_PER_BLOCK)
decompress_matvec_kernel(
    half *__restrict__ out,
    const uint4 *__restrict__ compressed,
    const half *__restrict__ codebook,
    const half2 *__restrict__ x,
    size_t iters_per_thread,
    size_t m,
    size_t n
) {
    const half *__restrict__ codebook_ptr = codebook;
    if constexpr (L < 16) {
        __shared__ half smem_codebook[1<<L];
        size_t offset_quarter = 1<<(L-2);
        // uint32_t *smem_x = reinterpret_cast<uint32_t *>(smem_codebook);
        // uint32_t *smem_y = reinterpret_cast<uint32_t *>(smem_codebook + offset_quarter);
        // uint32_t *smem_w = reinterpret_cast<uint32_t *>(smem_codebook + offset_quarter * 2);
        // uint32_t *smem_z = reinterpret_cast<uint32_t *>(smem_codebook + offset_quarter * 3);

        for (size_t idx = threadIdx.x; idx < (1<<L)/8; idx += blockDim.x) {
            // read in uint4 from global memory, and then write in uint32_t into shared memory
            uint4 quadruple = reinterpret_cast<const uint4 *>(codebook)[idx];
            reinterpret_cast<uint4 *>(smem_codebook)[idx] = quadruple;
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 1));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 2));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 3));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 4));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 5));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 6));
            // asm("prefetchu.L1 [%0];" : : "l" (codebook + idx * 8 + 7));
            // uintptr_t ptr = reinterpret_cast<uintptr_t>(&codebook[idx]);
            // if (ptr % 128 == 0) {
            //     asm("prefetch.local.L1 [%0];" : : "l" (ptr));
            // }
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 1));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 2));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 3));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 4));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 5));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 6));
            // asm("prefetch.global.L1 [%0];" : : "l" (codebook + idx * 8 + 7));
            // smem_x[idx] = quadruple.x;
            // smem_y[idx] = quadruple.y;
            // smem_z[idx] = quadruple.z;
            // smem_w[idx] = quadruple.w;
        }
        codebook_ptr = smem_codebook;
    }

    __syncthreads();

    constexpr uint16_t mask = (1<<L) - 1;
    constexpr uint16_t lane_mask = mask & ~0b111110;

    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    size_t laneId = threadIdx.x % warpSize;
    size_t warpId = threadId / warpSize;
    size_t strideC = blockDim.x * gridDim.x;
    size_t strideX = warpSize * 4;

    uint32_t carry = 0U;
    half2 inners[4] = {
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
    };

    for (size_t iter = 0; iter < iters_per_thread; iter += 1) {
        // uint4 elem = compressed[iter * strideC + threadId];

        uint4 elem;
        asm volatile ("ld.global.nc.L1::no_allocate.v4.u32 {%0,%1,%2,%3}, [%4];"
                : "=r"(elem.x), "=r"(elem.y), "=r"(elem.z), "=r"(elem.w)
                : "l" (compressed + iter * strideC + threadId));
        // ld.global.nc.v4.u32     {%r427, %r428, %r429, %r430}, [%rd323];

        // send w in lane X to carry in lane X+1, lane 0 not updated
        carry = __shfl_up_sync(FULL_MASK, elem.w, 1);

        // send w in lane 31 to carry in lane 0, lane 1-31 not updated
        uint32_t next_carry = __shfl_down_sync(FULL_MASK, elem.w, 31);

        uint32_t reg_c[5] = { carry, elem.x, elem.y, elem.z, elem.w };

        alias_half_uint16 reg_w[4][8];
        half2 reg_a[4][8];
        #pragma unroll
        for (size_t k = 0; k < 4; k += 1) {
            #pragma unroll
            for (size_t j = 0; j < 8; j += 1) {
                // TODO: would uint16_t be faster?
                uint32_t state_x = (lane_mask & __funnelshift_l(reg_c[k+1], reg_c[k], 4*j)) | (laneId << 1);
                uint32_t state_y = (lane_mask & __funnelshift_l(reg_c[k+1], reg_c[k], 4*j+2)) | (laneId << 1);

                // state_x = state_x * (2*state_x+1);
                // state_x = state_x * 1664525 + 1013904223;
                // state_y = state_y * (2*state_y+1);
                // state_y = state_y * 1664525 + 1013904223;

                // reg_w[k][j].foo = __halves2half2(codebook_ptr[idx_x], codebook_ptr[idx_y]);
                // TODO: Section 3.2 in https://ieeexplore.ieee.org/stamp/stamp.jsp?tp=&arnumber=10318209
                // reg_w[k][j].foo = h2cos(__halves2half2(
                //     __float2half(sinf(__uint_as_float(idx_x))),
                //     __float2half(fabsf(__uint_as_float(idx_y)))
                // ));
                asm volatile ("ld.global.nc.b16 %0, [%1];"
                        : "=h"(reg_w[k][j].bar[0])
                        : "l" (codebook_ptr + state_x));
                asm volatile ("ld.global.nc.b16 %0, [%1];"
                        : "=h"(reg_w[k][j].bar[1])
                        : "l" (codebook_ptr + state_y));
                reg_a[k][j] = x[((iter * 4 + k) * 8 + j) * warpSize + laneId];
            }
        }

        #pragma unroll
        for (size_t j = 0; j < 8; j += 1) {
            #pragma unroll
            for (size_t k = 0; k < 4; k += 1) {
                inners[k] = __hfma2(reg_w[k][j].foo, reg_a[k][j], inners[k]);
            }
        }

        carry = next_carry;
    }

    half2 inner01 = __hadd2(inners[0], inners[1]);
    half2 inner23 = __hadd2(inners[2], inners[3]);
    half2 inner0123 = __hadd2(inner01, inner23);

    for (size_t offset = 16; offset > 0; offset /= 2) {
        inner0123 = __hadd2(inner0123, __shfl_down_sync(FULL_MASK, inner0123, offset));
    }

    if (laneId == 0) {
        out[warpId] = __hadd(inner0123.x, inner0123.y);
    }
}


template <size_t L>
__host__ static float decompress_matvec(
    torch::Tensor &compressed,
    torch::Tensor &codebook,
    torch::Tensor &x,
    torch::Tensor &out
) {
    static_assert(L <= 32, "Shift register length should not exceed 32 as the kernel uses uint32_t");

    CHECK_INPUT(compressed);
    TORCH_CHECK(compressed.dim() == 3);
    TORCH_CHECK(compressed.size(2) == 32 * 4);  // each warp reads an uint4
    TORCH_CHECK(compressed.scalar_type() == torch::kInt32);

    size_t iters_per_thread = compressed.size(0);
    size_t m = compressed.size(1);
    size_t n = iters_per_thread * 32 * 4 * 16;

    CHECK_INPUT(codebook);
    TORCH_CHECK(codebook.dim() == 1);
    TORCH_CHECK(codebook.size(0) == 1<<L);
    TORCH_CHECK(codebook.scalar_type() == torch::kFloat16);

    CHECK_INPUT(x);
    TORCH_CHECK(x.dim() == 1);
    TORCH_CHECK(x.size(0) == n);
    TORCH_CHECK(x.scalar_type() == torch::kFloat16);

    CHECK_INPUT(out);
    TORCH_CHECK(out.dim() == 1);
    TORCH_CHECK(out.size(0) == m);
    TORCH_CHECK(out.scalar_type() == torch::kFloat16);

    size_t block_size = MAX_THREADS_PER_BLOCK;
    TORCH_CHECK(MAX_THREADS_PER_BLOCK % 32 == 0);
    size_t warps_per_block = MAX_THREADS_PER_BLOCK / 32;
    TORCH_CHECK(m % warps_per_block == 0);
    size_t grid_size = m / warps_per_block; // each warp takes care of a row

    gpuErrchk(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(
                decompress_matvec_kernel<L>),
                hipFuncAttributePreferredSharedMemoryCarveout,
                cudaSharedmemCarveoutMaxL1));

    hipStream_t stream;
    gpuErrchk(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipEventRecord(start, stream));

    decompress_matvec_kernel<L><<<grid_size, block_size>>>(
        (half *)out.data_ptr<c10::Half>(),
        (const uint4 *)compressed.data_ptr<int32_t>(),
        (const half *)codebook.data_ptr<c10::Half>(),
        (const half2 *)x.data_ptr<c10::Half>(),
        iters_per_thread,
        m,
        n);
    
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return msecTotal;
}

__host__ extern float decompress_matvec_16(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<16>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_14(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<14>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_12(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<12>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_10(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<10>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_8(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<8>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_6(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<6>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_4(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<4>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_2(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<2>(compressed, codebook, x, out);
}
