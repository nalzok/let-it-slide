#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <vector>
#include <utility>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include <cuda_pipeline.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace torch::indexing;
using namespace nvcuda;

#define MAX_THREADS_PER_BLOCK 256

#define FULL_MASK 0xFFFFFFFFU
#define HALF_MASK 0x0000FFFFU

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}


__global__ static void
__launch_bounds__(MAX_THREADS_PER_BLOCK)
decompress_kernel(
    half *__restrict__ out,
    const uint4 *__restrict__ compressed,
    const half *__restrict__ codebook,
    size_t compressed_m,
    size_t compressed_n
) {
    size_t laneId = threadIdx.x % warpSize;
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    for (int elem_idx = threadId; elem_idx < compressed_m * compressed_n; elem_idx += gridDim.x * blockDim.x) {
        uint4 inputs = compressed[elem_idx];
        uint16_t state = __shfl_up_sync(FULL_MASK, inputs.w, 1);
        if (laneId == 0) {
            if (elem_idx % compressed_n == 0) {
                // first element in a row
                state = 0;
            } else {
                const uint16_t *ptr = reinterpret_cast<const uint16_t *>(compressed + elem_idx);
                state = ptr[-1];
            }
        }

        half *__restrict__ output = out + elem_idx * 64;

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.x, state, 2);
            inputs.x <<= 2;
            output[i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.y, state, 2);
            inputs.y <<= 2;
            output[16+i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.z, state, 2);
            inputs.z <<= 2;
            output[32+i] = codebook[state];
        }

        #pragma unroll
        for (int i = 0; i < 16; i += 1) {
            state = __funnelshift_l(inputs.w, state, 2);
            inputs.w <<= 2;
            output[48+i] = codebook[state];
        }
    }
}


__host__ extern float decompress(
    torch::Tensor &compressed,
    torch::Tensor &codebook,
    torch::Tensor &out
) {
    CHECK_INPUT(compressed);
    TORCH_CHECK(compressed.dim() == 2);
    TORCH_CHECK(compressed.size(1) % (4 * 32) == 0);    // each warp has 32 threads, each handling an uint4
    TORCH_CHECK(compressed.scalar_type() == torch::kInt32);

    size_t compressed_m = compressed.size(0);
    size_t compressed_n = compressed.size(1) / 4;
    size_t m = compressed_m;
    size_t n = compressed_n * 64;   // at 2 bit, each uint4 has 4x32 bits = 4x16 weights

    CHECK_INPUT(codebook);
    TORCH_CHECK(codebook.dim() == 1);
    TORCH_CHECK(codebook.size(0) == 1<<16);
    TORCH_CHECK(codebook.scalar_type() == torch::kFloat16);

    CHECK_INPUT(out);
    TORCH_CHECK(out.dim() == 2);
    TORCH_CHECK(out.size(0) == m);
    TORCH_CHECK(out.size(1) == n);
    TORCH_CHECK(out.scalar_type() == torch::kFloat16);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, compressed.get_device());
    size_t grid_size = 2 * static_cast<size_t>(deviceProp.multiProcessorCount);
    size_t block_size = MAX_THREADS_PER_BLOCK;

    hipStream_t stream;
    gpuErrchk(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipEventRecord(start, stream));

    decompress_kernel<<<grid_size, block_size, 0, stream>>>(
        (half *)out.data_ptr<c10::Half>(),
        (const uint4 *)compressed.data_ptr<int32_t>(),
        (const half *)codebook.data_ptr<c10::Half>(),
        compressed_m,
        compressed_n);
    
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return msecTotal;
}


__global__ static void
__launch_bounds__(MAX_THREADS_PER_BLOCK)
surfaceWriteKernel(uint4 *gIData, hipSurfaceObject_t outputSurface) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    surf1Dwrite(gIData[idx], outputSurface, idx * sizeof(uint4));
}


template <size_t L>
__global__ static void
__launch_bounds__(MAX_THREADS_PER_BLOCK)
decompress_matvec_kernel(
    half *__restrict__ out,
    const uint4 *__restrict__ compressed,
    hipTextureObject_t codebook,
    const half2 *__restrict__ x,
    size_t iters_per_thread,
    size_t m,
    size_t n
) {
    constexpr uint16_t mask = (1<<L) - 1;

    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    size_t laneId = threadIdx.x % warpSize;
    size_t warpId = threadId / warpSize;
    size_t strideC = blockDim.x * gridDim.x;
    size_t strideX = warpSize * 4;

    uint32_t carry = 0U;
    half2 inners[4] = {
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
        __float2half2_rn(0.0f),
    };

    for (size_t iter = 0; iter < iters_per_thread; iter += 1) {
        uint4 elem = compressed[iter * strideC + threadId];

        // send w in lane X to carry in lane X+1, lane 0 not updated
        carry = __shfl_up_sync(FULL_MASK, elem.w, 1);

        // send w in lane 31 to carry in lane 0, lane 1-31 not updated
        uint32_t next_carry = __shfl_down_sync(FULL_MASK, elem.w, 31);

        uint32_t reg_c[5] = { carry, elem.x, elem.y, elem.z, elem.w };

        half2 reg_w[4][8];
        half2 reg_a[4][8];
        #pragma unroll
        for (size_t k = 0; k < 4; k += 1) {
            #pragma unroll
            for (size_t j = 0; j < 8; j += 1) {
                int16_t state_x = __funnelshift_l(reg_c[k+1], reg_c[k], 4*j);
                int16_t state_y = __funnelshift_l(reg_c[k+1], reg_c[k], 4*j+2);

                state_x = state_x * (2 * state_x + 1);
                state_x = state_x * 1664525 + 1013904223;
                state_y = state_y * (2 * state_y + 1);
                state_y = state_y * 1664525 + 1013904223;

                constexpr float converter = 1.f / INT16_MAX;
                reg_w[k][j] = __floats2half2_rn(
                    tex1D<float>(codebook, state_x*converter),
                    tex1D<float>(codebook, state_y*converter)
                );
                reg_a[k][j] = x[((iter * 4 + k) * 8 + j) * warpSize + laneId];
            }
        }

        #pragma unroll
        for (size_t j = 0; j < 8; j += 1) {
            #pragma unroll
            for (size_t k = 0; k < 4; k += 1) {
                inners[k] = __hfma2(reg_w[k][j], reg_a[k][j], inners[k]);
            }
        }

        carry = next_carry;
    }

    half2 inner01 = __hadd2(inners[0], inners[1]);
    half2 inner23 = __hadd2(inners[2], inners[3]);
    half2 inner0123 = __hadd2(inner01, inner23);

    for (size_t offset = 16; offset > 0; offset /= 2) {
        inner0123 = __hadd2(inner0123, __shfl_down_sync(FULL_MASK, inner0123, offset));
    }

    if (laneId == 0) {
        out[warpId] = __hadd(inner0123.x, inner0123.y);
    }
}


template <size_t L, size_t S>
__host__ static float decompress_matvec(
    torch::Tensor &compressed,
    torch::Tensor &codebook,
    torch::Tensor &x,
    torch::Tensor &out
) {
    static_assert(L <= 16, "Shift register length should not exceed 16 as the kernel uses int16_t");
    static_assert(S % 8 == 0, "Codebook size must be divisible by 8 as the kernel copies one uint4 at a time");

    CHECK_INPUT(compressed);
    TORCH_CHECK(compressed.dim() == 3);
    TORCH_CHECK(compressed.size(2) == 32 * 4);  // each warp reads an uint4
    TORCH_CHECK(compressed.scalar_type() == torch::kInt32);

    size_t iters_per_thread = compressed.size(0);
    size_t m = compressed.size(1);
    size_t n = iters_per_thread * 32 * 4 * 16;

    CHECK_INPUT(codebook);
    TORCH_CHECK(codebook.dim() == 1);
    TORCH_CHECK(codebook.size(0) == S);
    TORCH_CHECK(codebook.scalar_type() == torch::kFloat16);

    CHECK_INPUT(x);
    TORCH_CHECK(x.dim() == 1);
    TORCH_CHECK(x.size(0) == n);
    TORCH_CHECK(x.scalar_type() == torch::kFloat16);

    CHECK_INPUT(out);
    TORCH_CHECK(out.dim() == 1);
    TORCH_CHECK(out.size(0) == m);
    TORCH_CHECK(out.scalar_type() == torch::kFloat16);

    // copy codebook to a cuArray (texture data source) using surface writes
    hipChannelFormatDesc channelDesc = cudaCreateChannelDescHalf();
    hipArray *cuArray;
    hipExtent extent = { .width = S, .height = 0, .depth = 0 };
    gpuErrchk(hipMalloc3DArray(&cuArray, &channelDesc, extent, hipArraySurfaceLoadStore));

    hipSurfaceObject_t outputSurface;
    hipResourceDesc surfRes = {};
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = cuArray;

    gpuErrchk(hipCreateSurfaceObject(&outputSurface, &surfRes));
    surfaceWriteKernel<<<1, S/8>>>(reinterpret_cast<uint4 *>(codebook.data_ptr<c10::Half>()), outputSurface);

    hipTextureObject_t tex_codebook;

    hipResourceDesc texRes = {};
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = cuArray;

    hipTextureDesc texDescr = {};
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeMirror;
    texDescr.addressMode[1] = hipAddressModeMirror;
    texDescr.readMode = hipReadModeElementType;

    gpuErrchk(hipCreateTextureObject(&tex_codebook, &texRes, &texDescr, NULL));

    size_t block_size = MAX_THREADS_PER_BLOCK;
    TORCH_CHECK(MAX_THREADS_PER_BLOCK % 32 == 0);
    size_t warps_per_block = MAX_THREADS_PER_BLOCK / 32;
    TORCH_CHECK(m % warps_per_block == 0);
    size_t grid_size = m / warps_per_block; // each warp takes care of a row

    gpuErrchk(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(
                decompress_matvec_kernel<L>),
                hipFuncAttributePreferredSharedMemoryCarveout,
                cudaSharedmemCarveoutMaxL1));

    hipStream_t stream;
    gpuErrchk(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    gpuErrchk(hipStreamSynchronize(stream));
    gpuErrchk(hipEventRecord(start, stream));

    decompress_matvec_kernel<L><<<grid_size, block_size>>>(
        (half *)out.data_ptr<c10::Half>(),
        (const uint4 *)compressed.data_ptr<int32_t>(),
        tex_codebook,
        (const half2 *)x.data_ptr<c10::Half>(),
        iters_per_thread,
        m,
        n);
    
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    gpuErrchk(hipDestroySurfaceObject(outputSurface));
    gpuErrchk(hipDestroyTextureObject(tex_codebook));
    gpuErrchk(hipFreeArray(cuArray));

    return msecTotal;
}

__host__ extern float decompress_matvec_16_128(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<16, 128>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_16_64(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<16, 64>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_14_128(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<14, 128>(compressed, codebook, x, out);
}

__host__ extern float decompress_matvec_14_64(
    torch::Tensor &compressed, torch::Tensor &codebook, torch::Tensor &x, torch::Tensor &out
) {
    return decompress_matvec<14, 64>(compressed, codebook, x, out);
}
